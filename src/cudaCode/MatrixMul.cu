#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
template <size_t BLOCK_SIZE>
void __global__ MatMatMul(double *A,double *B,double *C,size_t m,size_t n,size_t p)
{
    
    double Cvalue = 0;  
    int row = blockIdx.y * blockDim.y + threadIdx.y;  
    int col = blockIdx.x * blockDim.x + threadIdx.x;  
    if(row<m&&col<p){
        for (int e = 0; e < n; ++e){
    		Cvalue += A[row * n+ e]* B[e * p+ col];  
    	}
       	C[row * p+ col] = Cvalue;
    }
}

extern "C"
void CpuMatriMul(char * h){
  	clock_t start = clock();
	int RowA=0; 
	int Col = 0;
	int ColB = 0;
	//��ȡ������״
	memcpy(&RowA, h, sizeof(int));
	memcpy(&Col, h+sizeof(int), sizeof(int));
	memcpy(&ColB, h+2*sizeof(int), sizeof(int));
	//�����ڴ�
	double *h_A,*h_B,*h_C;
	hipHostAlloc(&h_A,sizeof(double) * RowA* Col,hipHostMallocWriteCombined);
	hipHostAlloc(&h_B,sizeof(double) * Col * ColB ,hipHostMallocWriteCombined);
	hipHostAlloc(&h_C,sizeof(double) * RowA * ColB ,hipHostMallocWriteCombined);
	//h_A=(double *)malloc(RowA*Col*sizeof(double ));
	//h_B=(double *)malloc(ColB*Col*sizeof(double ));
	//h_C=(double *)malloc(RowA*ColB*sizeof(double ));
	char * h_a = h+3*sizeof(int);//A������ʼ��ַ
	char * h_b = h_a+RowA*Col*sizeof(double);   //B������ʼ��ַ
	//���ַ�����copy��double����
	for(int i=0;i<RowA*Col;i++){
		memcpy(h_A+i, h_a+i*sizeof(double),sizeof(double));
	}
	for(int i=0;i<ColB*Col;i++){
		memcpy(h_B+i, h_b+i*sizeof(double),sizeof(double));
	}	
	cout<<RowA<<Col<<ColB<<endl ;
	//CPU�汾
	clock_t cpustart  = clock();
   	for (int r = 0; r < RowA; r++) {
        	for (int c = 0; c < ColB; c++) {
            		for (int e = 0; e < Col; e++) {
                		h_C[r*ColB+ c] += h_A[r*Col+e] * h_B[e*ColB+c]; 
           		}
        	}
    	}
	
	clock_t cpuend  = clock();
	memcpy(h,&RowA,sizeof(int));
	memcpy(h+sizeof(int),&ColB,sizeof(int));
	h_a = h+2*sizeof(int);//A������ʼ��ַ
	for(int i=0;i<RowA*ColB;i++){
		memcpy(h_a+i*sizeof(double), h_C+i,sizeof(double));
	}
	hipHostFree(h_A);
	hipHostFree(h_B);
	hipHostFree(h_C);

	double cpulatency = (cpuend   - cpustart  )/ (double) CLOCKS_PER_SEC*1000 ;
	cout<<"CPU����ʱ��"<<cpulatency <<"ms"<<endl;
}

extern "C"
void matrixmul(char * h)
{
	clock_t start = clock();
	int RowA=0; 
	int Col = 0;
	int ColB = 0;
	//��ȡ������״
	memcpy(&RowA, h, sizeof(int));
	memcpy(&Col, h+sizeof(int), sizeof(int));
	memcpy(&ColB, h+2*sizeof(int), sizeof(int));
	//�����ڴ�
	double *h_A,*h_B,*h_C,*d_A,*d_B,*d_C;
	hipHostAlloc(&h_A,sizeof(double) * RowA* Col,hipHostMallocWriteCombined);
	hipHostAlloc(&h_B,sizeof(double) * Col * ColB ,hipHostMallocWriteCombined);
	hipHostAlloc(&h_C,sizeof(double) * RowA * ColB ,hipHostMallocWriteCombined);
	hipMalloc(&d_A,sizeof(double) * RowA* Col  );
	hipMalloc(&d_B,sizeof(double) * Col * ColB );
	hipMalloc(&d_C,sizeof(double) * RowA* ColB );
	//h_A=(double *)malloc(RowA*Col*sizeof(double ));
	//h_B=(double *)malloc(ColB*Col*sizeof(double ));
	//h_C=(double *)malloc(RowA*ColB*sizeof(double ));
	char * h_a = h+3*sizeof(int);//A������ʼ��ַ
	char * h_b = h_a+RowA*Col*sizeof(double);   //B������ʼ��ַ
	int count=0;
	//���ַ�����copy��double����
	for(int i=0;i<RowA*Col;i++){
		memcpy(h_A+i, h_a+i*sizeof(double),sizeof(double));
	}
	for(int i=0;i<ColB*Col;i++){
		memcpy(h_B+i, h_b+i*sizeof(double),sizeof(double));
	}	
	cout<<RowA<<Col<<ColB<<endl ;
	hipMemcpy(d_A,h_A,sizeof(double) * RowA* Col ,hipMemcpyHostToDevice);
	hipMemcpy(d_B,h_B,sizeof(double) * Col * ColB ,hipMemcpyHostToDevice);
	hipHostFree(h_A);
	hipHostFree(h_B);
	const dim3 gridDim((ColB+ 16- 1) / 16, (RowA+ 16- 1) / 16),blockDim(16, 16);

	MatMatMul<16><<<gridDim, blockDim>>>(d_A, d_B, d_C, RowA, Col, ColB);  
	hipDeviceSynchronize();
	hipMemcpy(h_C, d_C, ColB*RowA*sizeof(double), hipMemcpyDeviceToHost);
	memcpy(h,&RowA,sizeof(int));
	memcpy(h+sizeof(int),&ColB,sizeof(int));
	h_a = h+2*sizeof(int);//A������ʼ��ַ
	for(int i=0;i<RowA*ColB;i++){
		memcpy(h_a+i*sizeof(double), h_C+i,sizeof(double));
	}
	hipHostFree(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	double batchlatency = (clock() - start )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"GPU����ʱ��"<<batchlatency<<"ms"<<endl;

}