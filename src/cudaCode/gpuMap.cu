#include <hip/hip_runtime.h>
#include <pthread.h>
#include <unistd.h>
#include <iostream>
#include <list>
#include <vector>
//#include "../usecases/MapReduce.hpp"
//#include "../communication/Message.hpp"
//#include "../function/Function.hpp"
//#include "../partitioning/Partition.hpp"
//#include "../serialization/Serialization.hpp"

using namespace std;

__global__ void fuc(char *h, const int n){
	  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	 	    if (idx < n)
	    {
		
	    	int value=0;
		memcpy(&value, h+idx *4, sizeof(int));
		value= value*0.1;
		memcpy(h+idx *4, &value, sizeof(int));
	    	    }
}

void cpu(char *h, const size_t n){
	for(int i=0;i<n;i+=4){
		int value=0;
		memcpy(&value, h+i, sizeof(int));
		value= value*0.1;
		memcpy(h+i, &value, sizeof(int));
	}
}


int main(){
	clock_t start = clock();
	int nEle = 416000000;
	size_t nBytes = nEle*sizeof(int);
	int i=0;
	char * h,*d,*g_result;
	h=(char *)malloc(nBytes);
	hipMalloc((void **)&d, nBytes);
	g_result=(char *)malloc(nBytes);
	for(i=0;i<nBytes ;i+=4){
		memcpy(h+i, &i, sizeof(int));
	}
	cout<<endl;
	//for(i=0;i<nBytes ;i+=4){
	//	int value=0;
	//	memcpy(&value, h+i, sizeof(int));
	//	
	//}
	cout<<endl;
	cout<<endl;

	clock_t v2arr = clock();
	double translatency = (v2arr  - start )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"ת��������ʱ��"<<translatency<<"s"<<endl;
	cpu(h,nEle);
	clock_t cpu = clock();
	double cpulatency = (cpu  - v2arr )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"CPU����ʱ��"<<cpulatency <<"s"<<endl;
	for(i=0;i<416 ;i+=4){
		int value=0;
		memcpy(&value, h+i, sizeof(int));
		cout<<value;
	}
	
	for(i=0;i<nBytes ;i+=4){
		memcpy(h+i, &i, sizeof(int));
	}
	cout<<endl;	v2arr = clock();
	translatency = (v2arr  - cpu )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"ת��������ʱ��"<<translatency<<"s"<<endl;

	hipMemcpy(d, h, nBytes, hipMemcpyHostToDevice);
	clock_t copy = clock();
	double copylatency = (copy  - v2arr )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"H2D����ʱ��"<<copylatency<<"s"<<endl;

	int blocksize = 256;
	dim3 block (blocksize, 1);
	dim3 grid  ((nEle + block.x - 1) / block.x, 1);
	fuc<<<grid, block>>>(d, nEle);
	hipDeviceSynchronize();
	clock_t process = clock();
	double processlatency = (process - copy )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"kenel����ʱ��"<<processlatency <<"s"<<endl;

	hipMemcpy(g_result, d, nBytes, hipMemcpyDeviceToHost);
	for(i=0;i<416 ;i+=4){
		int value=0;
		memcpy(&value, g_result+i, sizeof(int));
		cout<<value;
	}
	cout<<endl;
	clock_t DTH = clock();
	double DTHlatency = (DTH  - process )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"DTH ����ʱ��"<<DTHlatency <<"s"<<endl;

	double batchlatency = (clock() - start )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"GPU����ʱ��"<<batchlatency<<"s"<<endl;

	return 0;

}
