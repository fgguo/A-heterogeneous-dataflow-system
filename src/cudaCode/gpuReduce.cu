#include <hip/hip_runtime.h>
#include <pthread.h>
#include <unistd.h>
#include <iostream>
#include <list>
#include <vector>
#include <stdio.h> 
#include <time.h>
//#include "../usecases/MapReduce.hpp"
//#include "../communication/Message.hpp"
//#include "../function/Function.hpp"
//#include "../partitioning/Partition.hpp"
//#include "../serialization/Serialization.hpp"

using namespace std;

__global__ void sum(char *g_idata, long long *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = 4*(blockIdx.x * blockDim.x + threadIdx.x);
    // convert global data pointer to the local pointer of this block
    char *idata = g_idata + 4*(blockIdx.x * blockDim.x);

    // boundary check
    if(idx >= n) return;
    long long v1,v2,v3;
    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
	    memcpy(&v1, idata+tid*4, sizeof(int));
	    memcpy(&v2, idata+(tid+stride)*4, sizeof(int));
	    v3 = v1+v2;
	    //printf("%d,%d,%d \n", v1,v2,v3);
	    memcpy(idata+tid*4,&v3,sizeof(int));
            //idata[tid] += idata[tid + stride];
        }

        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) 
	memcpy(g_odata+blockIdx.x,idata,sizeof(int));
}

long long cpu(char *h, const size_t n){
	long long sum=0;
	for(int i=0;i<n;i+=4){
		int value=0;
		memcpy(&value, h+i, sizeof(int));
		sum+= value;
	}
	return sum;
}


int main(){
	clock_t start = clock();
	int nEle = 416000000;
	size_t nBytes = nEle*sizeof(int);
	int i=0;
	char * h,*d;
        long long *g_result;
	h=(char *)malloc(nBytes);
	hipMalloc((void **)&d, nBytes);
	g_result=(long long *)malloc(nBytes);
	int m =0;
	for(i=0;i<nBytes ;i+=4){
		m=rand() % (100-10)+ 10;
		memcpy(h+i, &m, sizeof(int));
	}
	cout<<endl;
	//for(i=0;i<nBytes ;i+=4){
	//	int value=0;
	//	memcpy(&value, h+i, sizeof(int));
	//	
	//}
	cout<<endl;
	cout<<endl;

	clock_t v2arr = clock();
	double translatency = (v2arr  - start )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"ת��������ʱ��"<<translatency<<"s"<<endl;
	cout<<cpu(h,nBytes )<<endl;
	clock_t cpu = clock();
	double cpulatency = (cpu  - v2arr )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"CPU����ʱ��"<<cpulatency <<"ms"<<endl;
	v2arr = clock();
	translatency = (v2arr  - cpu )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"ת��������ʱ��"<<translatency<<"ms"<<endl;

	hipMemcpy(d, h, nBytes, hipMemcpyHostToDevice);
	clock_t copy = clock();
	double copylatency = (copy  - v2arr )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"H2D����ʱ��"<<copylatency<<"ms"<<endl;

	int blocksize = 256;
	dim3 block (blocksize, 1);
	dim3 grid  ((nEle + block.x - 1) / block.x, 1);
	long long *d_odata ;
        hipMalloc((void **) &d_odata, grid.x * sizeof(long long));
	sum<<<grid, block>>>(d,d_odata, nBytes );
	hipDeviceSynchronize();
	clock_t process = clock();
	double processlatency = (process - copy )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"kenel����ʱ��"<<processlatency <<"ms"<<endl;

	hipMemcpy(g_result, d_odata, grid.x * sizeof(long long), hipMemcpyDeviceToHost);
	long long gpu_sum = 0;
	
        for (int i = 0; i < grid.x; i++){
		gpu_sum+=g_result[i];
		}
	cout<<gpu_sum<<endl;
	clock_t DTH = clock();
	double DTHlatency = (DTH  - process )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"DTH ����ʱ��"<<DTHlatency <<"ms"<<endl;

	double batchlatency = (clock() - start )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"GPU����ʱ��"<<batchlatency<<"ms"<<endl;

	return 0;

}
