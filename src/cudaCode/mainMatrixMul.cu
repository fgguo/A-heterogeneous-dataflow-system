// CUDA runtime �� + CUBLAS ��
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include <iostream>
#include <stdlib.h>
#include "MatrixMul.cuh"
using namespace std;

// ������Ծ����ά��
int const A_ROW = 8192 ;
int const A_COL = 8192 ;
int const B_ROW = 8192 ;
int const B_COL = 8192 ;

extern "C"
void MatrixMul()
{
  // ����״̬����
  hipblasStatus_t status;
  float *h_A,*h_B,*h_C;   //�洢���ڴ��еľ���
  h_A = (float*)malloc(sizeof(float)*A_ROW*A_COL);  //���ڴ��п��ٿռ�
  h_B = (float*)malloc(sizeof(float)*B_ROW*B_COL);
  h_C = (float*)malloc(sizeof(float)*A_ROW*B_COL);

  // Ϊ����������Ԫ�ظ��� 0-10 ��Χ�ڵ������
  for (int i=0; i<A_ROW*A_COL; i++) {
    h_A[i] = (float)(rand()%10+1);
  }
  for(int i=0;i<B_ROW*B_COL; i++) {
    h_B[i] = (float)(rand()%10+1);
  }
  // ��ӡ�����Եľ���
  cout << "���� A :" << endl;
  for (int i=0; i<A_ROW*A_COL; i++){
    cout << h_A[i] << " ";
    if ((i+1)%A_COL == 0) cout << endl;
  }
  cout << endl;
  cout << "���� B :" << endl;
  for (int i=0; i<B_ROW*B_COL; i++){
    cout << h_B[i] << " ";
    if ((i+1)%B_COL == 0) cout << endl;
  }
  cout << endl;

  float *d_A,*d_B,*d_C;    //�洢���Դ��еľ���
  hipMalloc((void**)&d_A,sizeof(float)*A_ROW*A_COL); //���Դ��п��ٿռ�
  hipMalloc((void**)&d_B,sizeof(float)*B_ROW*B_COL);
  hipMalloc((void**)&d_C,sizeof(float)*A_ROW*B_COL);

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipMemcpy(d_A,h_A,sizeof(float)*A_ROW*A_COL,hipMemcpyHostToDevice); //���ݴ��ڴ濽�����Դ�
  hipMemcpy(d_B,h_B,sizeof(float)*B_ROW*B_COL,hipMemcpyHostToDevice);

  float a = 1, b = 0;
  hipblasSgemm(
          handle,
          HIPBLAS_OP_N,   //����A�����Բ�������ת�ã���������
          HIPBLAS_OP_N,   //����B�����Բ�������ת�ã���������
          B_COL,          //����B^T��C^T������
          A_ROW,          //����A^T��C^T������
          B_ROW,          //B^T��������A^T���������˴�Ҳ��ΪA_COL,һ����
          &a,             //alpha��ֵ
          d_B,            //�����ΪB^T
          B_COL,          //B^T��leading dimension���������ȣ���leading dimensionΪB^T������(B������)
          d_A,            //�Ҿ���ΪA^T
          A_COL,          //A^T��leading dimension���������ȣ���leading dimensionΪA^T������(A������)
          &b,             //beta��ֵ
          d_C,            //�������C
          B_COL           //C^T��leading dimension��C^T����һ���������ȣ���leading dimensionΪC^T������(C������)
  );
  //��ʱ�õ��Ľ������C=AB,������C�ǰ������ȣ��ʴ�ʱ�õ���CӦ������ȷ�����ת��
  std::cout << "��������ת�� ( (A*B)��ת�� )��" << std::endl;


  hipMemcpy(h_C,d_C,sizeof(float)*A_ROW*B_COL,hipMemcpyDeviceToHost);
  for(int i=0;i<A_ROW*B_COL;++i) {
    std::cout<<h_C[i]<<" ";
    if((i+1)%B_COL==0) std::cout<<std::endl;
  }
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(h_A);
  free(h_B);
  free(h_C);
}