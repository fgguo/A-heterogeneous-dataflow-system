#include <hip/hip_runtime.h>
#include <pthread.h>
#include <unistd.h>
#include <iostream>
#include <list>
#include <vector>
//#include "../usecases/MapReduce.hpp"
//#include "../communication/Message.hpp"
//#include "../function/Function.hpp"
//#include "../partitioning/Partition.hpp"
//#include "../serialization/Serialization.hpp"

using namespace std;

__global__ void fuc(char *h, const int n){
	  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	 	    if (idx < n)
	    {
		
	    	int value=0;
		memcpy(&value, h+idx *4, sizeof(int));
		value= value*0.01;
		memcpy(h+idx *4, &value, sizeof(int));
	    	    }
}

void cpu(char *h, const size_t n){
	for(int i=0;i<n;i+=4){
		int value=0;
		memcpy(&value, h+i, sizeof(int));
		value= value*0.01;
		memcpy(h+i, &value, sizeof(int));
	}
}

extern "C"
void mul(char *h,size_t nBytes ){
	clock_t start = clock();
	int i=0;
	char *d;
	hipMalloc((void **)&d, nBytes);	
	hipMemcpy(d, h, nBytes, hipMemcpyHostToDevice);
	clock_t copy = clock();
	double copylatency = (copy  - start )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"H2D����ʱ��"<<copylatency<<"s"<<endl;

	int blocksize = 256;
	dim3 block (blocksize, 1);
	dim3 grid  ((nBytes/4 + block.x - 1) / block.x, 1);
	fuc<<<grid, block>>>(d, nBytes /4);
	hipDeviceSynchronize();
	clock_t process = clock();
	double processlatency = (process - copy )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"kenel����ʱ��"<<processlatency <<"s"<<endl;

	hipMemcpy(h, d, nBytes, hipMemcpyDeviceToHost);
	cout<<endl;
	clock_t DTH = clock();
	double DTHlatency = (DTH  - process )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"DTH ����ʱ��"<<DTHlatency <<"s"<<endl;
	double batchlatency = (clock() - start )/ (double) CLOCKS_PER_SEC*1000 ; //batch latency calculation
	cout<<"GPU����ʱ��"<<batchlatency<<"s"<<endl;
}
